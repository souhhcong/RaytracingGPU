#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <vector>
#include <random>
#include <iostream>
#include <chrono>
#include <hiprand/hiprand_kernel.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define SQR(X) ((X)*(X))
#define NORMED_VEC(X) ((X) / (X).norm())
#ifndef PI
    #define PI 3.14159265358979323846
#endif
#define PRINT_VEC(v) (printf("%s: (%lf %lf %lf)\n", #v, (v)[0], (v)[1], (v)[2]))
#define INF (1e9+9)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ inline double uniform(hiprandState *rand_states, unsigned int tid) {
    hiprandState local_state = rand_states[tid];
    double RANDOM = hiprand_uniform( &local_state );
    rand_states[tid] = local_state;
	return RANDOM;
}

class Vector {
public:
	__device__ __host__ Vector(double x = 0, double y = 0, double z = 0) {
		data[0] = x;
		data[1] = y;
		data[2] = z;
	}
	__device__ __host__ double norm2() const {
		return data[0] * data[0] + data[1] * data[1] + data[2] * data[2];
	}
	__device__ __host__ double norm() const {
		return sqrt(norm2());
	}
	__device__ __host__ void normalize() {
		double n = norm();
		data[0] /= n;
		data[1] /= n;
		data[2] /= n;
	}
	__device__ __host__ double operator[](int i) const { return data[i]; };
	__device__ __host__ double& operator[](int i) { return data[i]; };
	double data[3];
};

__device__ __host__ Vector operator+(const Vector& a, const Vector& b) {
	return Vector(a[0] + b[0], a[1] + b[1], a[2] + b[2]);
}
__device__ __host__ Vector operator-(const Vector& a, const Vector& b) {
	return Vector(a[0] - b[0], a[1] - b[1], a[2] - b[2]);
}
__device__ __host__ Vector operator-(const Vector& a) {
	return Vector(-a[0], -a[1], -a[2]);
}
__device__ __host__ Vector operator*(const double a, const Vector& b) {
	return Vector(a*b[0], a*b[1], a*b[2]);
}
__device__ __host__ Vector operator*(const Vector& a, const double b) {
	return Vector(a[0]*b, a[1]*b, a[2]*b);
}
// Element wise vector multiplication
__device__ __host__ Vector operator*(const Vector& a, const Vector& b) {
	return Vector(a[0]*b[0], a[1]*b[1], a[2]*b[2]);
}
__device__ __host__ Vector operator/(const Vector& a, const double b) {
	return Vector(a[0] / b, a[1] / b, a[2] / b);
}
__device__ __host__ double dot(const Vector& a, const Vector& b) {
	return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}
__device__ __host__ Vector cross(const Vector& a, const Vector& b) {
	return Vector(a[1] * b[2] - a[2] * b[1], a[2] * b[0] - a[0] * b[2], a[0] * b[1] - a[1] * b[0]);
}

class Ray {
public:
	__device__ Ray(const Vector &O, const Vector &u, double refraction_index = 1.) : O(O), u(u), refraction_index(refraction_index) {};
	// ...
	Vector O, u;
	double refraction_index;
};

class Geometry {
public:
	__device__ Geometry(const Vector &C, double R, const Vector &albedo, int id = -1, bool mirror = 0, double in_refraction_index = 1, double out_refraction_index = 1): C(C), R(R), albedo(albedo), id(id),
	mirror(mirror), in_refraction_index(in_refraction_index), out_refraction_index(out_refraction_index) {}
	__device__ Geometry(): id(-1), mirror(0), in_refraction_index(1), out_refraction_index(1) {};
	
	Vector C;
    double R;
	Vector albedo;
	int id;
	bool mirror;
	double in_refraction_index;
	double out_refraction_index;
	
	__device__ bool intersect(const Ray &r, double &t, Vector &N) {
		double delta = SQR(dot(r.u, r.O - C)) - ((r.O - C).norm2() - R*R);
		if (delta < 0)
			return 0;
		double t1 = dot(r.u, C - r.O) - sqrt(delta); // first intersection
		double t2 = dot(r.u, C - r.O) + sqrt(delta); // second intersection
		if (t2 < 0)
			return 0;
		t = t1 < 0 ? t2 : t1;
		N = r.O + t * r.u - C;
		N.normalize();
		return 1;
	}
};

/* Start of code derived from Prof Bonnel's code */
class TriangleIndices {
public:
	__device__ __host__ TriangleIndices(int vtxi = -1, int vtxj = -1, int vtxk = -1, int ni = -1, int nj = -1, int nk = -1, int uvi = -1, int uvj = -1, int uvk = -1, int group = -1, bool added = false) : vtxi(vtxi), vtxj(vtxj), vtxk(vtxk), uvi(uvi), uvj(uvj), uvk(uvk), ni(ni), nj(nj), nk(nk), group(group) {
	};
	int vtxi, vtxj, vtxk; // indices within the vertex coordinates array
	int uvi, uvj, uvk;  // indices within the uv coordinates array
	int ni, nj, nk;  // indices within the normals array
	int group;       // face group
};

template <typename T> __device__ void swap ( T& a, T& b ) {
  T c(a); a=b; b=c;
}

class BoundingBox {
public:
	Vector mn, mx;

	__device__ BoundingBox(): mn(Vector(INF, INF, INF)), mx(Vector(-INF, -INF, -INF)) {};

	__device__ inline void update(const Vector &vec) {
		mn[0] = min(mn[0], vec[0]);
		mn[1] = min(mn[1], vec[1]);
		mn[2] = min(mn[2], vec[2]);
		mx[0] = max(mx[0], vec[0]);
		mx[1] = max(mx[1], vec[1]);
		mx[2] = max(mx[2], vec[2]);
	}

	__device__ inline bool intersect(const Ray &r, double &t) {
		double t0x = (mn[0] - r.O[0]) / r.u[0];
		double t0y = (mn[1] - r.O[1]) / r.u[1];
		double t0z = (mn[2] - r.O[2]) / r.u[2];
		double t1x = (mx[0] - r.O[0]) / r.u[0];
		double t1y = (mx[1] - r.O[1]) / r.u[1];
		double t1z = (mx[2] - r.O[2]) / r.u[2];
		if (t0x > t1x) swap(t0x, t1x);
		if (t0y > t1y) swap(t0y, t1y);
		if (t0z > t1z) swap(t0z, t1z);
		return min(t1x, min(t1y, t1z)) > max(t0x, max(t0y, t0z));
	}
};

class BVH {
public:
	BVH *left, *right;
	BoundingBox bb;
	int triangle_start, triangle_end;
};

class TriangleMeshHost {
public:
 	~TriangleMeshHost() {}
	TriangleMeshHost() {};

	void readOBJ(const char* obj) {

		char matfile[255];
		char grp[255];

		FILE* f;
		f = fopen(obj, "r");
		if (f == NULL) {
			printf("Error opening file!\n");
			return;
		}
		int curGroup = -1;
		while (!feof(f)) {
			char line[255];
			if (!fgets(line, 255, f)) break;

			std::string linetrim(line);
			linetrim.erase(linetrim.find_last_not_of(" \r\t") + 1);
			strcpy(line, linetrim.c_str());

			if (line[0] == 'u' && line[1] == 's') {
				sscanf(line, "usemtl %[^\n]\n", grp);
				curGroup++;
			}

			if (line[0] == 'v' && line[1] == ' ') {
				Vector vec;

				Vector col;
				if (sscanf(line, "v %lf %lf %lf %lf %lf %lf\n", &vec[0], &vec[1], &vec[2], &col[0], &col[1], &col[2]) == 6) {
					col[0] = std::min(1., std::max(0., col[0]));
					col[1] = std::min(1., std::max(0., col[1]));
					col[2] = std::min(1., std::max(0., col[2]));

					vertices.push_back(vec);
					vertexcolors.push_back(col);

				} else {
					sscanf(line, "v %lf %lf %lf\n", &vec[0], &vec[1], &vec[2]);
					vec = vec*0.8+Vector(0, -10, 0);
					vertices.push_back(vec);
				}
			}
			if (line[0] == 'v' && line[1] == 'n') {
				Vector vec;
				sscanf(line, "vn %lf %lf %lf\n", &vec[0], &vec[1], &vec[2]);
				normals.push_back(vec);
			}
			if (line[0] == 'v' && line[1] == 't') {
				Vector vec;
				sscanf(line, "vt %lf %lf\n", &vec[0], &vec[1]);
				uvs.push_back(vec);
			}
			if (line[0] == 'f') {
				TriangleIndices t;
				int i0, i1, i2, i3;
				int j0, j1, j2, j3;
				int k0, k1, k2, k3;
				int nn;
				t.group = curGroup;

				char* consumedline = line + 1;
				int offset;

				nn = sscanf(consumedline, "%u/%u/%u %u/%u/%u %u/%u/%u%n", &i0, &j0, &k0, &i1, &j1, &k1, &i2, &j2, &k2, &offset);
				if (nn == 9) {
					if (i0 < 0) t.vtxi = vertices.size() + i0; else	t.vtxi = i0 - 1;
					if (i1 < 0) t.vtxj = vertices.size() + i1; else	t.vtxj = i1 - 1;
					if (i2 < 0) t.vtxk = vertices.size() + i2; else	t.vtxk = i2 - 1;
					if (j0 < 0) t.uvi = uvs.size() + j0; else	t.uvi = j0 - 1;
					if (j1 < 0) t.uvj = uvs.size() + j1; else	t.uvj = j1 - 1;
					if (j2 < 0) t.uvk = uvs.size() + j2; else	t.uvk = j2 - 1;
					if (k0 < 0) t.ni = normals.size() + k0; else	t.ni = k0 - 1;
					if (k1 < 0) t.nj = normals.size() + k1; else	t.nj = k1 - 1;
					if (k2 < 0) t.nk = normals.size() + k2; else	t.nk = k2 - 1;
					indices.push_back(t);
				} else {
					nn = sscanf(consumedline, "%u/%u %u/%u %u/%u%n", &i0, &j0, &i1, &j1, &i2, &j2, &offset);
					if (nn == 6) {
						if (i0 < 0) t.vtxi = vertices.size() + i0; else	t.vtxi = i0 - 1;
						if (i1 < 0) t.vtxj = vertices.size() + i1; else	t.vtxj = i1 - 1;
						if (i2 < 0) t.vtxk = vertices.size() + i2; else	t.vtxk = i2 - 1;
						if (j0 < 0) t.uvi = uvs.size() + j0; else	t.uvi = j0 - 1;
						if (j1 < 0) t.uvj = uvs.size() + j1; else	t.uvj = j1 - 1;
						if (j2 < 0) t.uvk = uvs.size() + j2; else	t.uvk = j2 - 1;
						indices.push_back(t);
					} else {
						nn = sscanf(consumedline, "%u %u %u%n", &i0, &i1, &i2, &offset);
						if (nn == 3) {
							if (i0 < 0) t.vtxi = vertices.size() + i0; else	t.vtxi = i0 - 1;
							if (i1 < 0) t.vtxj = vertices.size() + i1; else	t.vtxj = i1 - 1;
							if (i2 < 0) t.vtxk = vertices.size() + i2; else	t.vtxk = i2 - 1;
							indices.push_back(t);
						} else {
							nn = sscanf(consumedline, "%u//%u %u//%u %u//%u%n", &i0, &k0, &i1, &k1, &i2, &k2, &offset);
							if (i0 < 0) t.vtxi = vertices.size() + i0; else	t.vtxi = i0 - 1;
							if (i1 < 0) t.vtxj = vertices.size() + i1; else	t.vtxj = i1 - 1;
							if (i2 < 0) t.vtxk = vertices.size() + i2; else	t.vtxk = i2 - 1;
							if (k0 < 0) t.ni = normals.size() + k0; else	t.ni = k0 - 1;
							if (k1 < 0) t.nj = normals.size() + k1; else	t.nj = k1 - 1;
							if (k2 < 0) t.nk = normals.size() + k2; else	t.nk = k2 - 1;
							indices.push_back(t);
						}
					}
				}

				consumedline = consumedline + offset;

				while (true) {
					if (consumedline[0] == '\n') break;
					if (consumedline[0] == '\0') break;
					nn = sscanf(consumedline, "%u/%u/%u%n", &i3, &j3, &k3, &offset);
					TriangleIndices t2;
					t2.group = curGroup;
					if (nn == 3) {
						if (i0 < 0) t2.vtxi = vertices.size() + i0; else	t2.vtxi = i0 - 1;
						if (i2 < 0) t2.vtxj = vertices.size() + i2; else	t2.vtxj = i2 - 1;
						if (i3 < 0) t2.vtxk = vertices.size() + i3; else	t2.vtxk = i3 - 1;
						if (j0 < 0) t2.uvi = uvs.size() + j0; else	t2.uvi = j0 - 1;
						if (j2 < 0) t2.uvj = uvs.size() + j2; else	t2.uvj = j2 - 1;
						if (j3 < 0) t2.uvk = uvs.size() + j3; else	t2.uvk = j3 - 1;
						if (k0 < 0) t2.ni = normals.size() + k0; else	t2.ni = k0 - 1;
						if (k2 < 0) t2.nj = normals.size() + k2; else	t2.nj = k2 - 1;
						if (k3 < 0) t2.nk = normals.size() + k3; else	t2.nk = k3 - 1;
						indices.push_back(t2);
						consumedline = consumedline + offset;
						i2 = i3;
						j2 = j3;
						k2 = k3;
					} else {
						nn = sscanf(consumedline, "%u/%u%n", &i3, &j3, &offset);
						if (nn == 2) {
							if (i0 < 0) t2.vtxi = vertices.size() + i0; else	t2.vtxi = i0 - 1;
							if (i2 < 0) t2.vtxj = vertices.size() + i2; else	t2.vtxj = i2 - 1;
							if (i3 < 0) t2.vtxk = vertices.size() + i3; else	t2.vtxk = i3 - 1;
							if (j0 < 0) t2.uvi = uvs.size() + j0; else	t2.uvi = j0 - 1;
							if (j2 < 0) t2.uvj = uvs.size() + j2; else	t2.uvj = j2 - 1;
							if (j3 < 0) t2.uvk = uvs.size() + j3; else	t2.uvk = j3 - 1;
							consumedline = consumedline + offset;
							i2 = i3;
							j2 = j3;
							indices.push_back(t2);
						} else {
							nn = sscanf(consumedline, "%u//%u%n", &i3, &k3, &offset);
							if (nn == 2) {
								if (i0 < 0) t2.vtxi = vertices.size() + i0; else	t2.vtxi = i0 - 1;
								if (i2 < 0) t2.vtxj = vertices.size() + i2; else	t2.vtxj = i2 - 1;
								if (i3 < 0) t2.vtxk = vertices.size() + i3; else	t2.vtxk = i3 - 1;
								if (k0 < 0) t2.ni = normals.size() + k0; else	t2.ni = k0 - 1;
								if (k2 < 0) t2.nj = normals.size() + k2; else	t2.nj = k2 - 1;
								if (k3 < 0) t2.nk = normals.size() + k3; else	t2.nk = k3 - 1;								
								consumedline = consumedline + offset;
								i2 = i3;
								k2 = k3;
								indices.push_back(t2);
							} else {
								nn = sscanf(consumedline, "%u%n", &i3, &offset);
								if (nn == 1) {
									if (i0 < 0) t2.vtxi = vertices.size() + i0; else	t2.vtxi = i0 - 1;
									if (i2 < 0) t2.vtxj = vertices.size() + i2; else	t2.vtxj = i2 - 1;
									if (i3 < 0) t2.vtxk = vertices.size() + i3; else	t2.vtxk = i3 - 1;
									consumedline = consumedline + offset;
									i2 = i3;
									indices.push_back(t2);
								} else {
									consumedline = consumedline + 1;
								}
							}
						}
					}
				}

			}

		}
		fclose(f);

	}

	std::vector<TriangleIndices> indices;
	std::vector<Vector> vertices;
	std::vector<Vector> normals;
	std::vector<Vector> uvs;
	std::vector<Vector> vertexcolors;
};

class Scene {
public:
	__device__ bool intersect_all(const Ray& r, Vector &P, Vector &N, int &objectId) {
		double t_min = INF;
		int id_min = -1;
		Vector N_min;
        for (int i = 0; i < objects_size; i++) {
            Geometry* object_ptr = objects[i];
			double t;
			double id = object_ptr->id;
			Vector N_tmp;
			bool ok = object_ptr->intersect(r, t, N_tmp);
			if (ok && t < t_min) {
				t_min = t;
				id_min = id;
				N_min = N_tmp;
			}
		}
		P = r.O + t_min * r.u;
		objectId = id_min;
		N = N_min;
		return id_min != -1;
	}

	__device__ Vector getColor(const Ray& ray, int ray_depth) {
		if (ray_depth < 0) return Vector(0., 0., 0.); // terminates recursion at some <- point
		Vector P, N;
		int sphere_id = -1;
		bool inter = intersect_all(ray, P, N, sphere_id);
		Vector color;
		if (inter) {
			if (objects[sphere_id]->mirror) {
				// Reflection
				double epsilon = 1e-6;
				Vector P_adjusted = P + epsilon * N;
				Vector new_direction = ray.u - 2 * dot(ray.u, N) * N;
				Ray reflected_ray(P_adjusted, new_direction, ray.refraction_index);
				return getColor(reflected_ray, ray_depth - 1);
			} else if (objects[sphere_id]->in_refraction_index != objects[sphere_id]->out_refraction_index) {
				// Refraction
				double epsilon = 1e-6;
				double refract_ratio;
				bool out2in = ray.refraction_index == objects[sphere_id]->out_refraction_index;
				if (out2in) { 
					// outside to inside
					refract_ratio = objects[sphere_id]->out_refraction_index / objects[sphere_id]->in_refraction_index;
				} else { 
					// inside to outside
					refract_ratio = objects[sphere_id]->in_refraction_index / objects[sphere_id]->out_refraction_index;
					N = -N;
				}
				if (((out2in && ray.refraction_index > objects[sphere_id]->in_refraction_index) ||
					(!out2in && ray.refraction_index > objects[sphere_id]->out_refraction_index)) &&
					SQR(refract_ratio) * (1 - SQR(dot(ray.u, N))) > 1) { 
					// total internal reflection
					return getColor(Ray(P + epsilon * N, ray.u - 2 * dot(ray.u, N) * N, ray.refraction_index), ray_depth - 1);
				}
				Vector P_adjusted = P - epsilon * N;
				Vector N_component = - sqrt(1 - SQR(refract_ratio) * (1 - SQR(dot(ray.u, N)))) * N;
				Vector T_component = refract_ratio * (ray.u - dot(ray.u, N) * N);
				Vector new_direction = N_component + T_component;
				if (out2in) {
					return getColor(Ray(P_adjusted, new_direction, objects[sphere_id]->in_refraction_index), ray_depth - 1);
				} else {
					return getColor(Ray(P_adjusted, new_direction, objects[sphere_id]->out_refraction_index), ray_depth - 1);
				}
			} else {
				// 	handle diffuse surfaces
				// 	Get shadow
				Vector P_prime;
				int sphere_id_shadow;
				double epsilon = 1e-6;
				Vector P_adjusted = P + epsilon * N;
				Vector direct_color, indirect_color;
				Vector N_prime;
				bool _ = intersect_all(Ray(P_adjusted, NORMED_VEC(L - P_adjusted)), P_prime, N_prime, sphere_id_shadow);
				
				if ((P_prime - P_adjusted).norm2() <= (L - P_adjusted).norm2()) {
					// Is shadow
					direct_color = Vector(0, 0, 0);
				} else {
					// Get direct color
					Geometry* S = objects[sphere_id];
					Vector wlight = L - P;
					wlight.normalize();
					double l = intensity / (4 * PI * (L - P).norm2()) * max(dot(N, wlight), 0.);
					direct_color = l * S->albedo / PI;
				}
				// Get indirect color by launching ray
				unsigned int seed = threadIdx.x;
				double r1 = uniform(rand_states, seed);
				double r2 = uniform(rand_states, seed);
				double x = cos(2 * PI * r1) * sqrt(1 - r2);
				double y = sin(2 * PI * r1) * sqrt(1 - r2);
				double z = sqrt(r2);
				Vector T1;
				if (abs(N[1]) != 0 && abs(N[0]) != 0) {
					T1 = Vector(-N[1], N[0], 0);
				} else {
					T1 = Vector(-N[2], 0, N[0]);
				}
				T1.normalize();
				Vector T2 = cross(N, T1);
				Vector random_direction = x * T1 + y * T2 + z * N;
				indirect_color = ((Geometry *)objects[sphere_id])->albedo * getColor(Ray(P_adjusted, random_direction), ray_depth - 1);
				color = direct_color + indirect_color;
			}
		}
		return color;
	}

	Geometry* objects[10];
    int objects_size = 0;
	double intensity = 3e10;
	Vector L;
	hiprandState* rand_states;
};

__global__ void KernelLaunch(double *colors, int W, int H, int num_rays, int num_bounce, TriangleIndices *indices, int indices_size, Vector *vertices, int vertices_size) {
	extern __shared__ double shared_memory[];
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
	double *shared_colors = shared_memory;
	Geometry *shared_objects = (Geometry *)&shared_colors[blockDim.x * 3];
	hiprandState *shared_rand_states = (hiprandState *)&shared_objects[10];
	Scene *shared_scene = (Scene *)&shared_rand_states[blockDim.x];
	if (!threadIdx.x) {
		shared_scene->L = Vector(-10., 20., 40.);
		shared_scene->objects_size = 0;
		shared_scene->intensity = 3e10;
		shared_objects[shared_scene->objects_size] = Geometry(Vector(0, 0, -1000), 940, Vector(0., 1., 0.));
		shared_objects[shared_scene->objects_size].id = shared_scene->objects_size;
		shared_scene->objects[shared_scene->objects_size] = &shared_objects[shared_scene->objects_size];
		++shared_scene->objects_size;
		shared_objects[shared_scene->objects_size] = Geometry(Vector(0, -1000, 0), 990, Vector(0., 0., 1.));
		shared_objects[shared_scene->objects_size].id = shared_scene->objects_size;
		shared_scene->objects[shared_scene->objects_size] = &shared_objects[shared_scene->objects_size];
		++shared_scene->objects_size;
		shared_objects[shared_scene->objects_size] = Geometry(Vector(0, 1000, 0), 940, Vector(1., 0., 0.));
		shared_objects[shared_scene->objects_size].id = shared_scene->objects_size;
		shared_scene->objects[shared_scene->objects_size] = &shared_objects[shared_scene->objects_size];
		++shared_scene->objects_size;
		shared_objects[shared_scene->objects_size] = Geometry(Vector(-1000, 0, 0), 940, Vector(0., 1., 1.));
		shared_objects[shared_scene->objects_size].id = shared_scene->objects_size;
		shared_scene->objects[shared_scene->objects_size] = &shared_objects[shared_scene->objects_size];
		++shared_scene->objects_size;
		shared_objects[shared_scene->objects_size] = Geometry(Vector(1000, 0, 0), 940, Vector(1., 1., 0.));
		shared_objects[shared_scene->objects_size].id = shared_scene->objects_size;
		shared_scene->objects[shared_scene->objects_size] = &shared_objects[shared_scene->objects_size];
		++shared_scene->objects_size;
		shared_objects[shared_scene->objects_size] = Geometry(Vector(0, 0, 1000), 940, Vector(1., 0., 1.));
		shared_objects[shared_scene->objects_size].id = shared_scene->objects_size;
		shared_scene->objects[shared_scene->objects_size] = &shared_objects[shared_scene->objects_size];
		++shared_scene->objects_size;
		shared_scene->rand_states = shared_rand_states;
	}
	__syncthreads();
	hiprand_init(123456, index, 0, shared_scene->rand_states + threadIdx.x);
    int i = (index / num_rays) / W, j = (index / num_rays) % W;
	Vector C(0, 0, 55);
	double alpha = PI/3;
	double z = -W / (2 * tan(alpha/2));
    unsigned int seed = threadIdx.x;
    Vector u_center((double)j - (double)W / 2 + 0.5, (double)H / 2 - i - 0.5, z);
	// Box-muller for anti-aliasing
	double sigma = 0.2;
	double r1 = uniform(shared_scene->rand_states, seed);
	double r2 = uniform(shared_scene->rand_states, seed);
	Vector u = u_center + Vector(sigma * sqrt(-2 * log(r1)) * cos(2 * PI * r2), sigma * sqrt(-2 * log(r1)) * sin(2 * PI * r2), 0);
	u.normalize();
	Ray r(C, u);
	Vector color = shared_scene->getColor(r, num_bounce);
	shared_colors[threadIdx.x * 3 + 0] = color[0];
    shared_colors[threadIdx.x * 3 + 1] = color[1];
    shared_colors[threadIdx.x * 3 + 2] = color[2];
	__syncthreads();
	colors[blockIdx.x * blockDim.x * 3 + blockDim.x * 0 + threadIdx.x] = shared_colors[blockDim.x * 0 + threadIdx.x];
	colors[blockIdx.x * blockDim.x * 3 + blockDim.x * 1 + threadIdx.x] = shared_colors[blockDim.x * 1 + threadIdx.x];
	colors[blockIdx.x * blockDim.x * 3 + blockDim.x * 2 + threadIdx.x] = shared_colors[blockDim.x * 2 + threadIdx.x];
}

int main(int argc, char **argv) {
    if (argc != 3) {
		std::cout << "Invalid number of arguments!\nThe first argument is number of rays and the second argument is number of bounces.\n";
		return 0;
	}
	auto start_time = std::chrono::system_clock::now();

	const int num_rays = atoi(argv[1]), num_bounce = atoi(argv[2]);
	int W = 512;
	int H = 512;
	int colors_size = sizeof(double) * H * W * 3 * num_rays;
	const int BLOCK_DIM = 128;
	int GRID_DIM = W * H * num_rays / BLOCK_DIM;
	
	Scene *d_s;
	double *h_colors, *d_colors;
    char *image;
	h_colors = new double[H * W * 3 * num_rays];
    image = new char[H * W * 3];

	// Increase stack size to 16KB per thread (Should be reduced in the future)
	gpuErrchk( hipDeviceSetLimit(hipLimitStackSize, 1<<14) );
	
	// Malloc & transfer to GPU
    gpuErrchk( hipMalloc((void**)&d_s, sizeof(Scene)) );
    gpuErrchk( hipMalloc((void**)&d_colors, colors_size) );
	TriangleMeshHost* mesh_ptr = new TriangleMeshHost(); // cat
	const char *path = "cadnav.com_model/Models_F0202A090/cat.obj";
	mesh_ptr->readOBJ(path);
	TriangleIndices* d_indices;
	Vector* d_vertices;
    gpuErrchk( hipMalloc((void**)&d_indices, mesh_ptr->indices.size() * sizeof(TriangleIndices)) );
    gpuErrchk( hipMemcpy(d_indices, &(mesh_ptr->indices[0]), mesh_ptr->indices.size() * sizeof(TriangleIndices), hipMemcpyHostToDevice) );
    gpuErrchk( hipMalloc((void**)&d_vertices, mesh_ptr->vertices.size() * sizeof(Vector)) );
    gpuErrchk( hipMemcpy(d_vertices, &(mesh_ptr->vertices[0]), mesh_ptr->vertices.size() * sizeof(Vector), hipMemcpyHostToDevice) );

    KernelLaunch<<<GRID_DIM, BLOCK_DIM, sizeof(double) * BLOCK_DIM * 3 + sizeof(Geometry) * 10 + sizeof(hiprandState) * BLOCK_DIM + sizeof(Scene)>>>(d_colors, W, H, num_rays, num_bounce, d_indices, mesh_ptr->indices.size(), d_vertices, mesh_ptr->vertices.size());
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipMemcpy(h_colors, d_colors, colors_size, hipMemcpyDeviceToHost) );
    gpuErrchk( hipFree(d_s) );
    gpuErrchk( hipFree(d_colors) );
    gpuErrchk( hipFree(d_indices) );
    gpuErrchk( hipFree(d_vertices) );

	for (int i = 0; i < H; ++i) {
		for (int j = 0; j < W; ++j) {
			Vector colors_sum;
			for (int t = 0; t < num_rays; ++t) {
				colors_sum = colors_sum + Vector(
					h_colors[((i * W + j) * num_rays + t) * 3 + 0],
					h_colors[((i * W + j) * num_rays + t) * 3 + 1],
					h_colors[((i * W + j) * num_rays + t) * 3 + 2]
				);
			}
			Vector colors_avg = colors_sum / num_rays;
			image[(i * W + j) * 3 + 0] = min(std::pow(colors_avg[0], 1./2.2), 255.);
			image[(i * W + j) * 3 + 1] = min(std::pow(colors_avg[1], 1./2.2), 255.);
			image[(i * W + j) * 3 + 2] = min(std::pow(colors_avg[2], 1./2.2), 255.);
		}
	}
	delete h_colors;
	stbi_write_png("image.png", W, H, 3, &image[0], 0);
    delete image;

    auto end_time = std::chrono::system_clock::now();
    std::chrono::duration<double> run_time = end_time-start_time;
    std::cout << "Rendering time: " << run_time.count() << " s\n";
}
