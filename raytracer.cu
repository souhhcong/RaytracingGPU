#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <vector>
#include <random>
#include <iostream>
#include <chrono>
#include <hiprand/hiprand_kernel.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define SQR(X) ((X)*(X))
#define NORMED_VEC(X) ((X) / (X).norm())
#ifndef PI
    #define PI 3.14159265358979323846
#endif
#define PRINT_VEC(v) (printf("%s: (%f %f %f)\n", #v, (v)[0], (v)[1], (v)[2]))
#define INF (1e9+9)
#define MAX_RAY_DEPTH 10
// #define float float

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ inline float uniform(hiprandState *rand_states, unsigned int tid) {
    hiprandState local_state = rand_states[tid];
 	float RANDOM = hiprand_uniform( &local_state );
    rand_states[tid] = local_state;
	return RANDOM;
}

class Vector {
public:
	__device__ __host__ Vector(float x = 0, float y = 0, float z = 0) {
		data[0] = x;
		data[1] = y;
		data[2] = z;
	}
	__device__ __host__ float norm2() const {
		return data[0] * data[0] + data[1] * data[1] + data[2] * data[2];
	}
	__device__ __host__ float norm() const {
		return sqrt(norm2());
	}
	__device__ __host__ void normalize() {
		float n = norm();
		data[0] /= n;
		data[1] /= n;
		data[2] /= n;
	}
	__device__ __host__ float operator[](int i) const { return data[i]; };
	__device__ __host__ float& operator[](int i) { return data[i]; };
	float data[3];
};

__device__ __host__ Vector operator+(const Vector& a, const Vector& b) {
	return Vector(a[0] + b[0], a[1] + b[1], a[2] + b[2]);
}
__device__ __host__ Vector operator-(const Vector& a, const Vector& b) {
	return Vector(a[0] - b[0], a[1] - b[1], a[2] - b[2]);
}
__device__ __host__ Vector operator-(const Vector& a) {
	return Vector(-a[0], -a[1], -a[2]);
}
__device__ __host__ Vector operator*(const float a, const Vector& b) {
	return Vector(a*b[0], a*b[1], a*b[2]);
}
__device__ __host__ Vector operator*(const Vector& a, const float b) {
	return Vector(a[0]*b, a[1]*b, a[2]*b);
}
// Element wise vector multiplication
__device__ __host__ Vector operator*(const Vector& a, const Vector& b) {
	return Vector(a[0]*b[0], a[1]*b[1], a[2]*b[2]);
}
__device__ __host__ Vector operator/(const Vector& a, const float b) {
	return Vector(a[0] / b, a[1] / b, a[2] / b);
}
__device__ __host__ float dot(const Vector& a, const Vector& b) {
	return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}
__device__ __host__ Vector cross(const Vector& a, const Vector& b) {
	return Vector(a[1] * b[2] - a[2] * b[1], a[2] * b[0] - a[0] * b[2], a[0] * b[1] - a[1] * b[0]);
}

class Ray {
public:
	__device__ Ray(const Vector &O, const Vector &u, float refraction_index = 1.) : O(O), u(u), refraction_index(refraction_index) {};
	// ...
	Vector O, u;
 float refraction_index;
};

class Geometry {
public:
	__device__ Geometry(const Vector &albedo, int id, bool mirror, float in_refraction_index, float out_refraction_index): albedo(albedo), id(id),
	mirror(mirror), in_refraction_index(in_refraction_index), out_refraction_index(out_refraction_index) {}
	__device__ Geometry(): mirror(0), in_refraction_index(1), out_refraction_index(1) {};

	Vector albedo;
	int id;
	bool mirror;
 float in_refraction_index;
 float out_refraction_index;
	__device__ virtual bool intersect(const Ray& r, float &t, Vector &N) { return 0; };
};

class Sphere: public Geometry {
public:
	__host__ __device__ Sphere(){};
	__device__ Sphere(const Vector &C, float R, const Vector& albedo, bool mirror = 0, float in_refraction_index = 1., float out_refraction_index = 1.) : 
	C(C), R(R), Geometry(albedo, id, mirror, in_refraction_index, out_refraction_index) {};
    Vector C;
 float R;
	__device__ bool intersect(const Ray &r, float &t, Vector &N) override {
	 float delta = SQR(dot(r.u, r.O - C)) - ((r.O - C).norm2() - R*R);
		if (delta < 0)
			return 0;
	 float t1 = dot(r.u, C - r.O) - sqrt(delta); // first intersection
	 float t2 = dot(r.u, C - r.O) + sqrt(delta); // second intersection
		if (t2 < 0)
			return 0;
		t = t1 < 0 ? t2 : t1;
		N = r.O + t * r.u - C;
		N.normalize();
		// printf("Intersect!\n");
		return 1;
	}
};

/* Start of code derived from Prof Bonnel's code */
class TriangleIndices {
public:
	__device__ __host__ TriangleIndices(int vtxi = -1, int vtxj = -1, int vtxk = -1, int ni = -1, int nj = -1, int nk = -1, int uvi = -1, int uvj = -1, int uvk = -1, int group = -1, bool added = false) : vtxi(vtxi), vtxj(vtxj), vtxk(vtxk), uvi(uvi), uvj(uvj), uvk(uvk), ni(ni), nj(nj), nk(nk), group(group){};
    int vtxi, vtxj, vtxk; // indices within the vertex coordinates array
    int uvi, uvj, uvk;    // indices within the uv coordinates array
    int ni, nj, nk;       // indices within the normals array
    int group;            // face group
};

template <typename T> __device__ __host__ void swap ( T& a, T& b ) {
  T c(a); a=b; b=c;
}

class BoundingBox {
public:
	Vector mn, mx;

	__device__ __host__ BoundingBox(): mn(Vector(INF, INF, INF)), mx(Vector(-INF, -INF, -INF)) {};
	__device__ __host__ BoundingBox(
		const Vector &mn_,
		const Vector &mx_
	) : mn(mn_),
		mx(mx_) {}

	__device__ __host__ inline void update(const Vector &vec) {
		mn[0] = min(mn[0], vec[0]);
		mn[1] = min(mn[1], vec[1]);
		mn[2] = min(mn[2], vec[2]);
		mx[0] = max(mx[0], vec[0]);
		mx[1] = max(mx[1], vec[1]);
		mx[2] = max(mx[2], vec[2]);
	}

	__device__ __host__ inline bool intersect(const Ray &r, float &t) {
	 float t0x = (mn[0] - r.O[0]) / r.u[0];
	 float t0y = (mn[1] - r.O[1]) / r.u[1];
	 float t0z = (mn[2] - r.O[2]) / r.u[2];
	 float t1x = (mx[0] - r.O[0]) / r.u[0];
	 float t1y = (mx[1] - r.O[1]) / r.u[1];
	 float t1z = (mx[2] - r.O[2]) / r.u[2];
	if (t0x > t1x) swap(t0x, t1x);
	if (t0y > t1y) swap(t0y, t1y);
	if (t0z > t1z) swap(t0z, t1z);

	// printf("%f %f %f", t0x, t0y, t0z)
	// PRINT_VEC(mn);
	// PRINT_VEC(mx);

	return min(t1x, min(t1y, t1z)) > max(t0x, max(t0y, t0z));
	}
};

class BVH {
public:
	BVH *left, *right;
	BoundingBox bb;
	int triangle_start, triangle_end;
};

class BVHDevice {
public:
	int left, right;
	BoundingBox bb;
	int triangle_start, triangle_end;
};

class TriangleMesh: public Geometry {
public:
  	// __device__ ~TriangleMesh() {};
	__device__ TriangleMesh() {};

	#define between(A, B, C) ((A) <= (B) && (B) <= (C))

	__device__ void get_smooth_normal(Ray r, TriangleIndices tid, Vector &N){
		Vector A, B, C;
		float alpha, t;

		A = vertices[tid.vtxi];
		B = vertices[tid.vtxj];
		C = vertices[tid.vtxk];

		Vector e1 = B - A;
		Vector e2 = C - A;
		N = cross(e1, e2);
		float beta = dot(e2, cross(A - r.O, r.u)) / dot(r.u, N);
		float gamma = - dot(e1, cross(A - r.O, r.u)) / dot(r.u, N);
		t = dot(A - r.O, N) / dot(r.u, N);
	
		alpha = 1 - beta - gamma;
		Vector Na, Nb, Nc;
		// printf("%d %d %d\n", tid.ni, tid.nj, tid.nk);
		Na = normals[tid.ni];
		Nb = normals[tid.nj];
		Nc = normals[tid.nk];
		N = alpha * Na + beta * Nb + gamma * Nc;
		// PRINT_VEC(Na);
		N.normalize();
	}

	__device__ bool moller_trumbore(const Vector &A, const Vector &B, const Vector &C, Vector& N, const Ray &r, float &t) {
		Vector e1 = B - A;
		Vector e2 = C - A;
		N = cross(e1, e2);
		if (dot(r.u, N) == 0) return 0;
		float beta = dot(e2, cross(A - r.O, r.u)) / dot(r.u, N);
		float gamma = - dot(e1, cross(A - r.O, r.u)) / dot(r.u, N);
		if (!between(0, beta, 1) || !between(0, gamma, 1))	return 0;
		t = dot(A - r.O, N) / dot(r.u, N);
		return beta + gamma <= 1 && t > 0;
	}
	
	__device__ bool intersect(const Ray &r, float &t, Vector &N) override {
	 float t_tmp;

		// #define BUILD_BVH(var, idx) var.left = bvh[(idx) * 10 + 0],\
		// 							var.right = bvh[(idx) * 10 + 1],\
		// 							var.bb = BoundingBox(\
		// 								Vector(\
		// 									bvh[(idx) * 10 + 2],\
		// 									bvh[(idx) * 10 + 3],\
		// 									bvh[(idx) * 10 + 4]\
		// 								),\
		// 								Vector(\
		// 									bvh[(idx) * 10 + 5],\
		// 									bvh[(idx) * 10 + 6],\
		// 									bvh[(idx) * 10 + 7]\
		// 								)\
		// 							),\
		// 							var.triangle_start = bvh[(idx) * 10 + 8],\
		// 							var.triangle_end = bvh[(idx) * 10 + 9]
		// PRINT_VEC(tmp);
		BVH root_bvh = bvh;
		// BUILD_BVH(root_bvh, 0);
		if (!root_bvh.bb.intersect(r, t_tmp)) {
			return 0;
		}

		BVH* s[30];
		int s_size = 0;
		s[s_size++] = &root_bvh;


	 float t_min = INF;
	 int idx_min = -1;
		while (s_size) {
			BVH *cur = s[s_size-1];
			s_size--;
			// BVHDevice cur_bvh;
			// BUILD_BVH(cur_bvh, cur);
			if (cur->left != NULL) {
				// BVHDevice left_bvh;
				// BUILD_BVH(left_bvh, cur.left);
				// BVHDevice right_bvh;
				// BUILD_BVH(right_bvh, cur.right);
			 float t_left, t_right;
				bool ok_left = cur->left->bb.intersect(r, t_left);
				bool ok_right = cur->right->bb.intersect(r, t_right);
				if (ok_left) s[s_size++] = cur->left;
				if (ok_right) s[s_size++] = cur->right;
			} else {
				// Leaf
				for (int i = cur->triangle_start; i < cur->triangle_end; i++) {
				 float t_cur;
					Vector A = vertices[indices[i].vtxi], B = vertices[indices[i].vtxj], C = vertices[indices[i].vtxk];
					Vector N_triangle;
					bool inter = moller_trumbore(A, B, C, N_triangle, r, t_cur);
					if (!inter) continue;
					if (t_cur > 1e-4f && t_cur < t_min) {
						t_min = t_cur;
						N = N_triangle;
						idx_min = i;
						//
						// PRINT_VEC(N);
					}
				} 
			}
		}
		N.normalize();
		if(idx_min > -1)
			// PRINT_VEC(N);
			get_smooth_normal(r, indices[idx_min], N);
			// printf("new N ");
			// PRINT_VEC(N);
		t = t_min;
		if(t_min != INF){
			// printf("inter triangle %f\n", t_min);
		}else{
			printf("no hit\n");
		}
		return t_min != INF;
	}

	__device__	BoundingBox compute_bbox(int triangle_start, int triangle_end) {
		BoundingBox bb;
		for (int i = triangle_start; i < triangle_end; i++) {
			bb.update(vertices[indices[i].vtxi]);
			bb.update(vertices[indices[i].vtxj]);
			bb.update(vertices[indices[i].vtxk]);
		}
		return bb;
	}

	__device__ void buildBVH(BVH* cur, int triangle_start, int triangle_end) {
		// std::cout << cur << ' ' << triangle_start << ' ' << triangle_end << '\n';
		// printf("%d %d\n", triangle_start, triangle_end);
		cur->triangle_start = triangle_start;
		cur->triangle_end = triangle_end;
		cur->left = NULL;
		cur->right = NULL;
		cur->bb = compute_bbox(triangle_start, triangle_end);

		Vector diag = cur->bb.mx - cur->bb.mn;
		int max_axis;
		if (diag[0] >= diag[1] && diag[0] >= diag[2])
			max_axis = 0;
		else if (diag[1] >= diag[0] && diag[1] >= diag[2])
			max_axis = 1;
		else
			max_axis = 2;

		int pivot = triangle_start;
	 	float split = (cur->bb.mn[max_axis] + cur->bb.mx[max_axis]) / 2;
		for (int i = triangle_start; i < triangle_end; i++) {
		 float cen = (vertices[indices[i].vtxi][max_axis] + vertices[indices[i].vtxj][max_axis] + vertices[indices[i].vtxk][max_axis]) / 3;
			if (cen < split) {
				swap(indices[i], indices[pivot]);
				pivot++;
			}
		}

		if (pivot <= triangle_start || pivot >= triangle_end - 1 || triangle_end - triangle_start < 5) {
			return;
		}
		cur->left = new BVH;
		cur->right = new BVH;
		buildBVH(cur->left, triangle_start, pivot);
		buildBVH(cur->right, pivot, triangle_end);
	}

	__device__ void bvhTreeToArray(BVH *cur, float* bvh_arr, size_t &arr_size, size_t arr_idx = 0) {
		// std::cout << arr_idx << ' ' << cur->triangle_start << ' ' << cur->triangle_end << '\n';
		// std::cout << "rfgsg\n";
		// PRINT_VEC(cur->bb.mn);
		// PRINT_VEC(cur->bb.mx);
		
		bvh_arr[arr_idx * 10 + 2] = cur->bb.mn[0];
		bvh_arr[arr_idx * 10 + 3] = cur->bb.mn[1];
		bvh_arr[arr_idx * 10 + 4] = cur->bb.mn[2];
		bvh_arr[arr_idx * 10 + 5] = cur->bb.mx[0];
		bvh_arr[arr_idx * 10 + 6] = cur->bb.mx[1];
		bvh_arr[arr_idx * 10 + 7] = cur->bb.mx[2];
		bvh_arr[arr_idx * 10 + 8] = cur->triangle_start;
		bvh_arr[arr_idx * 10 + 9] = cur->triangle_end;

		if (cur->left) {
			bvh_arr[arr_idx * 10 + 0] = arr_size++;
			bvhTreeToArray(cur->left, bvh_arr, arr_size, bvh_arr[arr_idx * 10 + 0]);
		} else {
			bvh_arr[arr_idx * 10 + 0] = -1;
		}
		if (cur->right) {
			bvh_arr[arr_idx * 10 + 1] = arr_size++;
			bvhTreeToArray(cur->right, bvh_arr, arr_size, bvh_arr[arr_idx * 10 + 1]);
		} else {
			bvh_arr[arr_idx * 10 + 1] = -1;
		}
	}

	TriangleIndices* indices;
	int indices_size;
	Vector* vertices, *normals;
	int vertices_size, normals_size;
	// float* bvh;
	BVH bvh;
};

__device__ Vector rotate(const Vector &v, const float *R) {
    return Vector(
        R[0] * v[0] + R[1] * v[1] + R[2] * v[2],
        R[3] * v[0] + R[4] * v[1] + R[5] * v[2],
        R[6] * v[0] + R[7] * v[1] + R[8] * v[2]
    );
}

__global__ void transform(Vector *vertices, int vertices_size, Vector *normals, int normals_size, Vector translation, const float *rotation_matrix) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < vertices_size) {
        // Transform the vertex
        vertices[idx] = rotate(vertices[idx], rotation_matrix);
        vertices[idx][0] += translation[0];
        vertices[idx][1] += translation[1];
        vertices[idx][2] += translation[2];
    }

    if (idx < normals_size) {
        // Transform the normal
        normals[idx] = rotate(normals[idx], rotation_matrix);
		normals[idx][0] += translation[0];
        normals[idx][1] += translation[1];
        normals[idx][2] += translation[2];
    }
}

class TriangleMeshHost {
public:
 	~TriangleMeshHost() {}
	TriangleMeshHost() {};
	void rescale(float scale, Vector offset){
		for(int i = 0; i < vertices.size(); i++){
			vertices[i] = vertices[i] * scale + offset;
		}
	}

	
void readOBJ(const char *obj)
{

    char matfile[255];
    char grp[255];

    FILE *f;
    f = fopen(obj, "r");
    int curGroup = -1;
    while (!feof(f))
    {
        char line[255];
        if (!fgets(line, 255, f))
            break;

        std::string linetrim(line);
        linetrim.erase(linetrim.find_last_not_of(" \r\t") + 1);
        strcpy(line, linetrim.c_str());

        if (line[0] == 'u' && line[1] == 's')
        {
            sscanf(line, "usemtl %[^\n]\n", grp);
            curGroup++;
        }

        if (line[0] == 'v' && line[1] == ' ')
        {
            Vector vec;

            Vector col;
            if (sscanf(line, "v %f %f %f %f %f %f\n", &vec[0], &vec[1], &vec[2], &col[0], &col[1], &col[2]) == 6)
            {
                col[0] = std::min(1.f, std::max(0.f, col[0]));
                col[1] = std::min(1.f, std::max(0.f, col[1]));
                col[2] = std::min(1.f, std::max(0.f, col[2]));

                vertices.push_back(vec);
                vertexcolors.push_back(col);
            }
            else
            {
                sscanf(line, "v %f %f %f\n", &vec[0], &vec[1], &vec[2]);
                vertices.push_back(vec);
            }
        }
        if (line[0] == 'v' && line[1] == 'n')
        {
            Vector vec;
            sscanf(line, "vn %f %f %f\n", &vec[0], &vec[1], &vec[2]);
            normals.push_back(vec);
        }
        if (line[0] == 'v' && line[1] == 't')
        {
            Vector vec;
            sscanf(line, "vt %f %f\n", &vec[0], &vec[1]);
            uvs.push_back(vec);
        }
        if (line[0] == 'f')
        {
            TriangleIndices t;
            int i0, i1, i2, i3;
            int j0, j1, j2, j3;
            int k0, k1, k2, k3;
            int nn;
            t.group = curGroup;

            char *consumedline = line + 1;
            int offset;

            nn = sscanf(consumedline, "%u/%u/%u %u/%u/%u %u/%u/%u%n", &i0, &j0, &k0, &i1, &j1, &k1, &i2, &j2, &k2, &offset);
            if (nn == 9)
            {
                if (i0 < 0)
                    t.vtxi = vertices.size() + i0;
                else
                    t.vtxi = i0 - 1;
                if (i1 < 0)
                    t.vtxj = vertices.size() + i1;
                else
                    t.vtxj = i1 - 1;
                if (i2 < 0)
                    t.vtxk = vertices.size() + i2;
                else
                    t.vtxk = i2 - 1;
                if (j0 < 0)
                    t.uvi = uvs.size() + j0;
                else
                    t.uvi = j0 - 1;
                if (j1 < 0)
                    t.uvj = uvs.size() + j1;
                else
                    t.uvj = j1 - 1;
                if (j2 < 0)
                    t.uvk = uvs.size() + j2;
                else
                    t.uvk = j2 - 1;
                if (k0 < 0)
                    t.ni = normals.size() + k0;
                else
                    t.ni = k0 - 1;
                if (k1 < 0)
                    t.nj = normals.size() + k1;
                else
                    t.nj = k1 - 1;
                if (k2 < 0)
                    t.nk = normals.size() + k2;
                else
                    t.nk = k2 - 1;
                indices.push_back(t);
            }
            else
            {
                nn = sscanf(consumedline, "%u/%u %u/%u %u/%u%n", &i0, &j0, &i1, &j1, &i2, &j2, &offset);
                if (nn == 6)
                {
                    if (i0 < 0)
                        t.vtxi = vertices.size() + i0;
                    else
                        t.vtxi = i0 - 1;
                    if (i1 < 0)
                        t.vtxj = vertices.size() + i1;
                    else
                        t.vtxj = i1 - 1;
                    if (i2 < 0)
                        t.vtxk = vertices.size() + i2;
                    else
                        t.vtxk = i2 - 1;
                    if (j0 < 0)
                        t.uvi = uvs.size() + j0;
                    else
                        t.uvi = j0 - 1;
                    if (j1 < 0)
                        t.uvj = uvs.size() + j1;
                    else
                        t.uvj = j1 - 1;
                    if (j2 < 0)
                        t.uvk = uvs.size() + j2;
                    else
                        t.uvk = j2 - 1;
                    indices.push_back(t);
                }
                else
                {
                    nn = sscanf(consumedline, "%u %u %u%n", &i0, &i1, &i2, &offset);
                    if (nn == 3)
                    {
                        if (i0 < 0)
                            t.vtxi = vertices.size() + i0;
                        else
                            t.vtxi = i0 - 1;
                        if (i1 < 0)
                            t.vtxj = vertices.size() + i1;
                        else
                            t.vtxj = i1 - 1;
                        if (i2 < 0)
                            t.vtxk = vertices.size() + i2;
                        else
                            t.vtxk = i2 - 1;
                        indices.push_back(t);
                    }
                    else
                    {
                        nn = sscanf(consumedline, "%u//%u %u//%u %u//%u%n", &i0, &k0, &i1, &k1, &i2, &k2, &offset);
                        if (i0 < 0)
                            t.vtxi = vertices.size() + i0;
                        else
                            t.vtxi = i0 - 1;
                        if (i1 < 0)
                            t.vtxj = vertices.size() + i1;
                        else
                            t.vtxj = i1 - 1;
                        if (i2 < 0)
                            t.vtxk = vertices.size() + i2;
                        else
                            t.vtxk = i2 - 1;
                        if (k0 < 0)
                            t.ni = normals.size() + k0;
                        else
                            t.ni = k0 - 1;
                        if (k1 < 0)
                            t.nj = normals.size() + k1;
                        else
                            t.nj = k1 - 1;
                        if (k2 < 0)
                            t.nk = normals.size() + k2;
                        else
                            t.nk = k2 - 1;
                        indices.push_back(t);
                    }
                }
            }

            consumedline = consumedline + offset;

            while (true)
            {
                if (consumedline[0] == '\n')
                    break;
                if (consumedline[0] == '\0')
                    break;
                nn = sscanf(consumedline, "%u/%u/%u%n", &i3, &j3, &k3, &offset);
                TriangleIndices t2;
                t2.group = curGroup;
                if (nn == 3)
                {
                    if (i0 < 0)
                        t2.vtxi = vertices.size() + i0;
                    else
                        t2.vtxi = i0 - 1;
                    if (i2 < 0)
                        t2.vtxj = vertices.size() + i2;
                    else
                        t2.vtxj = i2 - 1;
                    if (i3 < 0)
                        t2.vtxk = vertices.size() + i3;
                    else
                        t2.vtxk = i3 - 1;
                    if (j0 < 0)
                        t2.uvi = uvs.size() + j0;
                    else
                        t2.uvi = j0 - 1;
                    if (j2 < 0)
                        t2.uvj = uvs.size() + j2;
                    else
                        t2.uvj = j2 - 1;
                    if (j3 < 0)
                        t2.uvk = uvs.size() + j3;
                    else
                        t2.uvk = j3 - 1;
                    if (k0 < 0)
                        t2.ni = normals.size() + k0;
                    else
                        t2.ni = k0 - 1;
                    if (k2 < 0)
                        t2.nj = normals.size() + k2;
                    else
                        t2.nj = k2 - 1;
                    if (k3 < 0)
                        t2.nk = normals.size() + k3;
                    else
                        t2.nk = k3 - 1;
                    indices.push_back(t2);
                    consumedline = consumedline + offset;
                    i2 = i3;
                    j2 = j3;
                    k2 = k3;
                }
                else
                {
                    nn = sscanf(consumedline, "%u/%u%n", &i3, &j3, &offset);
                    if (nn == 2)
                    {
                        if (i0 < 0)
                            t2.vtxi = vertices.size() + i0;
                        else
                            t2.vtxi = i0 - 1;
                        if (i2 < 0)
                            t2.vtxj = vertices.size() + i2;
                        else
                            t2.vtxj = i2 - 1;
                        if (i3 < 0)
                            t2.vtxk = vertices.size() + i3;
                        else
                            t2.vtxk = i3 - 1;
                        if (j0 < 0)
                            t2.uvi = uvs.size() + j0;
                        else
                            t2.uvi = j0 - 1;
                        if (j2 < 0)
                            t2.uvj = uvs.size() + j2;
                        else
                            t2.uvj = j2 - 1;
                        if (j3 < 0)
                            t2.uvk = uvs.size() + j3;
                        else
                            t2.uvk = j3 - 1;
                        consumedline = consumedline + offset;
                        i2 = i3;
                        j2 = j3;
                        indices.push_back(t2);
                    }
                    else
                    {
                        nn = sscanf(consumedline, "%u//%u%n", &i3, &k3, &offset);
                        if (nn == 2)
                        {
                            if (i0 < 0)
                                t2.vtxi = vertices.size() + i0;
                            else
                                t2.vtxi = i0 - 1;
                            if (i2 < 0)
                                t2.vtxj = vertices.size() + i2;
                            else
                                t2.vtxj = i2 - 1;
                            if (i3 < 0)
                                t2.vtxk = vertices.size() + i3;
                            else
                                t2.vtxk = i3 - 1;
                            if (k0 < 0)
                                t2.ni = normals.size() + k0;
                            else
                                t2.ni = k0 - 1;
                            if (k2 < 0)
                                t2.nj = normals.size() + k2;
                            else
                                t2.nj = k2 - 1;
                            if (k3 < 0)
                                t2.nk = normals.size() + k3;
                            else
                                t2.nk = k3 - 1;
                            consumedline = consumedline + offset;
                            i2 = i3;
                            k2 = k3;
                            indices.push_back(t2);
                        }
                        else
                        {
                            nn = sscanf(consumedline, "%u%n", &i3, &offset);
                            if (nn == 1)
                            {
                                if (i0 < 0)
                                    t2.vtxi = vertices.size() + i0;
                                else
                                    t2.vtxi = i0 - 1;
                                if (i2 < 0)
                                    t2.vtxj = vertices.size() + i2;
                                else
                                    t2.vtxj = i2 - 1;
                                if (i3 < 0)
                                    t2.vtxk = vertices.size() + i3;
                                else
                                    t2.vtxk = i3 - 1;
                                consumedline = consumedline + offset;
                                i2 = i3;
                                indices.push_back(t2);
                            }
                            else
                            {
                                consumedline = consumedline + 1;
                            }
                        }
                    }
                }
            }
        }
    }
    fclose(f);
};

	std::vector<TriangleIndices> indices;
	std::vector<Vector> vertices;
	std::vector<Vector> normals;
	std::vector<Vector> uvs;
	std::vector<Vector> vertexcolors;
	BVH bvh;
	size_t n_bvhs = 0;
	#define between(A, B, C) ((A) <= (B) && (B) <= (C))


};

class Scene {
public:
	__device__ void addObject(Geometry* s) {
		s->id = objects_size;
		objects[objects_size++] = s;
	}

	__device__ bool intersect_all(const Ray& r, Vector &P, Vector &N, int &objectId) {
	 float t_min = INF;
		int id_min = -1;
		Vector N_min;
        for (int i = 0; i < objects_size; i++) {
            Geometry* object_ptr = objects[i];
		 float t;
		 float id = object_ptr->id;
			Vector N_tmp;
			bool ok = object_ptr->intersect(r, t, N_tmp);
			if (ok && t < t_min) {
				t_min = t;
				id_min = id;
				N_min = N_tmp;
			}
		}
		P = r.O + t_min * r.u;
		objectId = id_min;
		N = N_min;
		return id_min != -1;
	}

	__device__ Vector getColorIterative(hiprandState *rand_state, const Ray& input_ray, int max_ray_depth) {
		int types[MAX_RAY_DEPTH];
		Vector direct_colors[MAX_RAY_DEPTH];
		Vector indirect_albedos[MAX_RAY_DEPTH];
		Ray ray = input_ray;
		for (int ray_depth = 0; ray_depth < max_ray_depth; ray_depth++) {
			Vector P, N;
			int sphere_id = -1;
			bool inter = intersect_all(ray, P, N, sphere_id);
			Vector color;
			if (inter) {
				if (objects[sphere_id]->mirror) {
					// Reflection
					types[ray_depth] = 0;
				 float epsilon = 1e-4;
					Vector P_adjusted = P + epsilon * N;
					Vector new_direction = ray.u - 2 * dot(ray.u, N) * N;
					Ray reflected_ray(P_adjusted, new_direction, ray.refraction_index);
					ray = reflected_ray;
				} else if (objects[sphere_id]->in_refraction_index != objects[sphere_id]->out_refraction_index) {
					// Refraction
					types[ray_depth] = 0;
				 float epsilon = 1e-4;
				 float refract_ratio;
					bool out2in = ray.refraction_index == objects[sphere_id]->out_refraction_index;
					if (out2in) { 
						// outside to inside
						refract_ratio = objects[sphere_id]->out_refraction_index / objects[sphere_id]->in_refraction_index;
					} else { 
						// inside to outside
						refract_ratio = objects[sphere_id]->in_refraction_index / objects[sphere_id]->out_refraction_index;
						N = -N;
					}
					if (((out2in && ray.refraction_index > objects[sphere_id]->in_refraction_index) ||
						(!out2in && ray.refraction_index > objects[sphere_id]->out_refraction_index)) &&
						SQR(refract_ratio) * (1 - SQR(dot(ray.u, N))) > 1) { 
						// total internal reflection
						ray = Ray(P + epsilon * N, ray.u - 2 * dot(ray.u, N) * N, ray.refraction_index);
						continue;
					}
					Vector P_adjusted = P - epsilon * N;
					Vector N_component = - sqrt(1 - SQR(refract_ratio) * (1 - SQR(dot(ray.u, N)))) * N;
					Vector T_component = refract_ratio * (ray.u - dot(ray.u, N) * N);
					Vector new_direction = N_component + T_component;
					if (out2in) {
						ray = Ray(P_adjusted, new_direction, objects[sphere_id]->in_refraction_index);
					} else {
						ray = Ray(P_adjusted, new_direction, objects[sphere_id]->out_refraction_index);
					}
				} else {
					// 	handle diffuse surfaces
					// 	Get shadow
					Vector P_prime;
					int sphere_id_shadow;
				 	float epsilon = 1e-4;
					Vector P_adjusted = P + epsilon * N;
					Vector N_prime;
					bool _ = intersect_all(Ray(P_adjusted, NORMED_VEC(L - P_adjusted)), P_prime, N_prime, sphere_id_shadow);
					
					if ((P_prime - P_adjusted).norm2() <= (L - P_adjusted).norm2()) {
						// Is shadow
						direct_colors[ray_depth] = Vector(0.f, 0.f, 0.f);
					} else {
						// Get direct color
						Geometry* S = objects[sphere_id];
						Vector wlight = L - P;
						wlight.normalize();
					 float l = intensity / (4 * PI * (L - P).norm2()) * max(dot(N, wlight), 0.f);
						direct_colors[ray_depth] = l * S->albedo / PI;
					}
					// printf("%f %f\n", (P_prime - P_adjusted).norm2(), (L - P_adjusted).norm2());
					// Get indirect color by launching ray
					unsigned int seed = threadIdx.x;
					float r1 = hiprand_uniform(rand_state);
					float r2 = hiprand_uniform(rand_state);
					float x = cos(2 * PI * r1) * sqrt(1 - r2);
					float y = sin(2 * PI * r1) * sqrt(1 - r2);
					float z = sqrt(r2);
					Vector T1;
					if (abs(N[1]) != 0.f && abs(N[0]) != 0.f) {
						T1 = Vector(-N[1], N[0], 0);
					} else {
						T1 = Vector(-N[2], 0, N[0]);
					}
					T1.normalize();
					Vector T2 = cross(N, T1);
					Vector random_direction = x * T1 + y * T2 + z * N;
					ray = Ray(P_adjusted, random_direction);
					indirect_albedos[ray_depth] = ((Geometry *)objects[sphere_id])->albedo;
					types[ray_depth] = 1;

					// printf("DIFF INTER\n");

				}
			}
		}
		Vector ans_color;
		for (int i = max_ray_depth - 1; i >= 0; i--) {
			if (types[i]) {
				// Hits a diffusion object
				ans_color = indirect_albedos[i] * ans_color + direct_colors[i];
			}
		}
		
		return ans_color;
	}

	Geometry* objects[10];
    int objects_size = 0;
 	float intensity = 3e10;
	Vector L;
	hiprandState* rand_states;
};

__global__ void KernelInit(Scene *s, TriangleIndices *indices, int indices_size, Vector *vertices, int vertices_size,Vector *normals, int normals_size) {
 	int threadId = threadIdx.x + blockIdx.x * blockDim.x;
	if (!threadId) {
		s->L = Vector(-10., 20., 40.);
		s->objects_size = 0;
		s->intensity = 3e10;
		// s->addObject(new Sphere(Vector(0, 0, 0), 10, Vector(1., 1., 1.))); // white sphere
		s->addObject(new Sphere(Vector(0, 0, -1000), 940.0f, Vector(0.0f, 1.0f, 0.0f))); // green fore wall
		s->addObject(new Sphere(Vector(0, -1000, 0), 990.0f, Vector(0.0f, 0.0f, 1.0f))); // blue floor
		s->addObject(new Sphere(Vector(0, 1000, 0), 940.0f, Vector(1.0f, 0.0f, 0.0f))); // red ceiling
		s->addObject(new Sphere(Vector(-1000, 0, 0), 940.0f, Vector(0.0f, 1.0f, 1.0f))); // cyan left wall
		s->addObject(new Sphere(Vector(1000, 0, 0), 940.0f, Vector(1.0f, 1.0f, 0.0f))); // yellow right wall
		s->addObject(new Sphere(Vector(0, 0, 1000), 940.0f, Vector(1.0f, 0.0f, 1.0f))); // magenta back wall
		// s->addObject(new Sphere(Vector(-20, 0, 0), 10, Vector(0., 0., 0.), 1)); // mirror sphere
		// s->addObject(new Sphere(Vector(20, 0, 0), 9, Vector(0., 0., 0.), 0, 1, 1.5)); // inner nested ssphere
		// s->addObject(new Sphere(Vector(20, 0, 0), 10, Vector(0., 0., 0.), 0, 1.5, 1)); // outer nested sphere

		TriangleMesh* cat = new TriangleMesh();
		cat->albedo = Vector(0.25f, 0.25f, 0.25f);
	 	cat->indices_size = indices_size;
		cat->indices = indices;
		cat->vertices_size = vertices_size;
		cat->vertices = vertices;
		// printf("%d %d\n", indices_size, vertices_size);
		cat->normals_size = normals_size;
		cat->normals = normals;
		// for(int i = 0; i < 10; i++){
		// 	PRINT_VEC(cat->normals[i]);
		// }
		// cat->uvs_size;
		// cat->uvs;
		// cat->vertexcolors_size;
		// cat->vertexcolors;
		cat->bvh.bb = cat->compute_bbox(0, cat->indices_size);
		cat->buildBVH(&(cat->bvh), 0, cat->indices_size);
		s->addObject(cat);
	}
}

__global__ void KernelLaunch(Scene *s, Vector *colors, int W, int H, int num_rays, int num_bounce) {
    // size_t index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprandState rand_state; // state of the random number generator, to prevent repetition
	hiprand_init(threadId, 0, 0, &rand_state);

	Vector outcolor;
	int i = y*W + x; // pixel index in buffer
	// float coordx = (float) x / W; // pixel x-coordinate on screen
	// int coordy = (float) y / H;

	outcolor = Vector(0.f, 0.f, 0.f);
	
	// hiprand_init(123456, index, 0, shared_scene->rand_states + threadIdx.x);
    // int i = (index / num_rays) / W, j = (index / num_rays) % W;
	
	// printf("%d %d\n", x, y);
	Vector C(0, 0, 55);
	float alpha = PI/3;
	float z = -W / (2 * tan(alpha/2));

    Vector u_center(x -  (float)W / 2 + 0.5,  (float)H / 2 - y - 0.5, z);
	// Box-muller for anti-aliasing

	float sigma = 0.2;
	for(int i = 0; i < num_rays; i++){
		float r1 = hiprand_uniform(&rand_state);
		float r2 = hiprand_uniform(&rand_state);
		Vector u = u_center + Vector(sigma * sqrt(-2 * log(r1)) * cos(2 * PI * r2), sigma * sqrt(-2 * log(r1)) * sin(2 * PI * r2), 0);
		u.normalize();
		Ray r(C, u);
		Vector color = s->getColorIterative(&rand_state, r, num_bounce);
		outcolor = outcolor + color;
	}
	// PRINT_VEC(color);
	outcolor = outcolor / num_rays;
	colors[i] = outcolor;	
}

void allocateAndCopyDataToDevice(TriangleMeshHost* mesh_ptr, Vector*& d_vertices, Vector*& d_normals, TriangleIndices*& d_indices) {
    gpuErrchk(hipMalloc((void**)&d_vertices, mesh_ptr->vertices.size() * sizeof(Vector)));
    gpuErrchk(hipMemcpy(d_vertices, &(mesh_ptr->vertices[0]), mesh_ptr->vertices.size() * sizeof(Vector), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void**)&d_normals, mesh_ptr->normals.size() * sizeof(Vector)));
    gpuErrchk(hipMemcpy(d_normals, &(mesh_ptr->normals[0]), mesh_ptr->normals.size() * sizeof(Vector), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void**)&d_indices, mesh_ptr->indices.size() * sizeof(TriangleIndices)));
    gpuErrchk(hipMemcpy(d_indices, &(mesh_ptr->indices[0]), mesh_ptr->indices.size() * sizeof(TriangleIndices), hipMemcpyHostToDevice));
}

void transformMesh(Vector* d_vertices, int vertices_size, Vector* d_normals, int normals_size, const Vector& translation, const float* rotation_matrix) {
    float* d_rotation_matrix;
    gpuErrchk(hipMalloc(&d_rotation_matrix, 9 * sizeof(float)));
    gpuErrchk(hipMemcpy(d_rotation_matrix, rotation_matrix, 9 * sizeof(float), hipMemcpyHostToDevice));

    const int threadsPerBlock = 256;
    const int max_size = max(vertices_size, normals_size);
    const int numBlocks = (max_size + threadsPerBlock - 1) / threadsPerBlock;

    transform<<<numBlocks, threadsPerBlock>>>(d_vertices, vertices_size, d_normals, normals_size, translation, d_rotation_matrix);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    hipFree(d_rotation_matrix);
}

void renderScene(Scene* d_s, Vector* d_colors, int W, int H, int num_rays, int num_bounce) {
    dim3 block(16, 16, 1);
    dim3 grid(W / block.x, H / block.y, 1);

    KernelLaunch<<<grid, block>>>(d_s, d_colors, W, H, num_rays, num_bounce);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
}

void saveImage(Vector* h_colors, int W, int H) {
    char* image = new char[W * H * 3];
    for (int i = 0; i < H; ++i) {
        for (int j = 0; j < W; ++j) {
            image[(i * W + j) * 3 + 0] = min(pow(h_colors[(i * W + j)][0], 1.0 / 2.2), 255.0);
            image[(i * W + j) * 3 + 1] = min(pow(h_colors[(i * W + j)][1], 1.0 / 2.2), 255.0);
            image[(i * W + j) * 3 + 2] = min(pow(h_colors[(i * W + j)][2], 1.0 / 2.2), 255.0);
        }
    }
    stbi_write_png("image.png", W, H, 3, image, 0);
    delete[] image;
}

int main(int argc, char **argv) {
    if (argc != 3) {
		std::cout << "Invalid number of arguments!\nThe first argument is number of rays and the second argument is number of bounces.\n";
		return 0;
	}

	/*
		Measure runtime
	*/
	auto start_time = std::chrono::system_clock::now();

	const int num_rays = atoi(argv[1]), num_bounce = atoi(argv[2]);
	int W = 512;
	int H = 512;
	int colors_size = sizeof(float) * H * W * 3 * num_rays;
	const int BLOCK_DIM = 128;
	int GRID_DIM = W * H * num_rays / BLOCK_DIM;
	float angle = -M_PI/3;

    Vector translation = {0.f, 0.f, 0.f};
    // float rotation_matrix[9] = {
    //     cos(angle), -sin(angle), 0.,
    //     sin(angle), cos(angle), 0.,
    //     0., 0., 1.
    // };
	float rotation_matrix[9] = {
        cos(angle), 0, sin(angle),
       	0, 1, 0,
        -sin(angle), 0., cos(angle),
    };
	
	Scene *d_s;
	Vector *h_colors, *d_colors;
    char *image;
	h_colors = new Vector[H * W];
    image = new char[H * W * 3];

	gpuErrchk( hipDeviceSetLimit(hipLimitStackSize, 1<<14) );
	
	// Malloc & transfer to GPU
    gpuErrchk( hipMalloc((void**)&d_s, sizeof(Scene)) );
    gpuErrchk( hipMalloc((void**)&d_colors, H * W * sizeof(Vector)) );

	/*
		Instantiate cat object
	*/
	TriangleMeshHost* mesh_ptr = new TriangleMeshHost(); // cat
	const char *path = "cadnav.com_model/Models_F0202A090/cat.obj";
	mesh_ptr->readOBJ(path);
	mesh_ptr->rescale(0.6f, Vector(0.f, -10.f, 0.f));

	/*
		Transfer remaining neccessary mesh information to GPU
	*/
	Vector* d_vertices;
    Vector* d_normals;
    TriangleIndices* d_indices;

    allocateAndCopyDataToDevice(mesh_ptr, d_vertices, d_normals, d_indices);

	float *d_rotation_matrix;
	hipMalloc(&d_rotation_matrix, 9 * sizeof(float));
	hipMemcpy(d_rotation_matrix, rotation_matrix, 9 * sizeof(float), hipMemcpyHostToDevice);

	transformMesh(d_vertices, mesh_ptr->vertices.size(), d_normals, mesh_ptr->normals.size(), translation, rotation_matrix);

	KernelInit<<<1, 1>>>(d_s, d_indices, mesh_ptr->indices.size(), d_vertices, mesh_ptr->vertices.size(), d_normals, mesh_ptr->normals.size());

	dim3 block(16, 16, 1);
	dim3 grid(W / block.x, H / block.y, 1);

	renderScene(d_s, d_colors, W, H, num_rays, num_bounce);

	/*
		Transfer result back from GPU
		Clean memory
		Deduce final result
	*/
    gpuErrchk( hipMemcpy(h_colors, d_colors, H * W * sizeof(Vector), hipMemcpyDeviceToHost) );
    gpuErrchk( hipFree(d_s) );
    gpuErrchk( hipFree(d_colors) );
    gpuErrchk( hipFree(d_indices) );
    gpuErrchk( hipFree(d_vertices) );
	// delete[] arr_bvh;
	saveImage(h_colors, W, H);
    delete[] h_colors;

	/*
		Measure runtime
	*/
    auto end_time = std::chrono::system_clock::now();
    std::chrono::duration<double> run_time = end_time - start_time;
    std::cout << "Rendering time: " << run_time.count() << " s\n";

	return 0;
}
